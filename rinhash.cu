#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdint.h>
#include <stdio.h>
#include <string.h>
#include <vector>
#include <stdexcept>

// Include shared device functions
#include "rinhash_device.cuh"
#include "argon2d_device.cuh"
#include "sha3-256.cu"
#include "blake3_device.cuh"


// External references to our CUDA implementations
extern "C" void blake3_hash(const uint8_t* input, size_t input_len, uint8_t* output);
extern "C" void argon2d_hash_rinhash(uint8_t* output, const uint8_t* input, size_t input_len);
extern "C" void sha3_256_hash(const uint8_t* input, size_t input_len, uint8_t* output);

// Modified kernel to use device functions
extern "C" __global__ void rinhash_cuda_kernel(
    const uint8_t* input, 
    size_t input_len, 
    uint8_t* output
) {
    // Intermediate results in shared memory
    __shared__ uint8_t blake3_out[32];
    __shared__ uint8_t argon2_out[32];
    // Only one thread should do this work
    if (threadIdx.x == 0) {
        // Step 1: BLAKE3 hash - now using light_hash_device
        light_hash_device(input, input_len, blake3_out);
        // Step 2: Argon2d hash
        uint32_t m_cost = 64000; // Example
        size_t memory_size = m_cost * sizeof(block);
        block* d_memory = (block*)malloc(memory_size);
        uint8_t salt[11] = { 'R','i','n','C','o','i','n','S','a','l','t' };
        device_argon2d_hash(argon2_out, blake3_out, 32, 2, 64000, 1, d_memory, salt, 11);
        
        // Step 3: SHA3-256 hash
        uint8_t sha3_out[32];
        sha3_256_device(argon2_out, 32, sha3_out);
        
    }
    
    // Use syncthreads to ensure all threads wait for the computation to complete
    __syncthreads();
}

// RinHash CUDA implementation
extern "C" void rinhash_cuda(const uint8_t* input, size_t input_len, uint8_t* output) {
    // Allocate device memory
    uint8_t *d_input = nullptr;
    uint8_t *d_output = nullptr;

    hipError_t err;

    // Allocate memory on device
    err = hipMalloc(&d_input, input_len);
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: Failed to allocate input memory: %s\n", hipGetErrorString(err));
        return;
    }

    err = hipMalloc(&d_output, 32);
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: Failed to allocate output memory: %s\n", hipGetErrorString(err));
        hipFree(d_input);
        return;
    }

    // Copy input data to device
    err = hipMemcpy(d_input, input, input_len, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: Failed to copy input to device: %s\n", hipGetErrorString(err));
        hipFree(d_input);
        hipFree(d_output);
        return;
    }

    // Launch the kernel
    rinhash_cuda_kernel<<<1, 1>>>(d_input, input_len, d_output);

    // Wait for kernel to finish
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error during kernel execution: %s\n", hipGetErrorString(err));
        hipFree(d_input);
        hipFree(d_output);
        return;
    }

    // Copy result back to host
    err = hipMemcpy(output, d_output, 32, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: Failed to copy output from device: %s\n", hipGetErrorString(err));
    }

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
}


// Helper function to convert a block header to bytes
extern "C" void blockheader_to_bytes(
    const uint32_t* version,
    const uint32_t* prev_block,
    const uint32_t* merkle_root,
    const uint32_t* timestamp,
    const uint32_t* bits,
    const uint32_t* nonce,
    uint8_t* output,
    size_t* output_len
) {
    size_t offset = 0;
    
    // Version (4 bytes)
    memcpy(output + offset, version, 4);
    offset += 4;
    
    // Previous block hash (32 bytes)
    memcpy(output + offset, prev_block, 32);
    offset += 32;
    
    // Merkle root (32 bytes)
    memcpy(output + offset, merkle_root, 32);
    offset += 32;
    
    // Timestamp (4 bytes)
    memcpy(output + offset, timestamp, 4);
    offset += 4;
    
    // Bits (4 bytes)
    memcpy(output + offset, bits, 4);
    offset += 4;
    
    // Nonce (4 bytes)
    memcpy(output + offset, nonce, 4);
    offset += 4;
    
    *output_len = offset;
}

// Batch processing version for mining
extern "C" void rinhash_cuda_batch(
    const uint8_t* block_headers,
    size_t block_header_len,
    uint8_t* outputs,
    uint32_t num_blocks
) {
    // Reset device to clear any previous errors
    hipError_t err = hipDeviceReset();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: Failed to reset device: %s\n", 
                hipGetErrorString(err));
        return;
    }
    
    
    // Check available memory
    size_t free_mem, total_mem;
    err = hipMemGetInfo(&free_mem, &total_mem);
    if (err != hipSuccess) {
        //fprintf(stderr, "CUDA error: Failed to get memory info: %s\n", 
        //        hipGetErrorString(err));
        return;
    }
    
    size_t headers_size = num_blocks * block_header_len;
    size_t outputs_size = num_blocks * 32;
    size_t required_mem = headers_size + outputs_size;
    
    if (required_mem > free_mem) {
        fprintf(stderr, "CUDA error: Not enough memory (required: %zu, free: %zu)\n", 
                required_mem, free_mem);
        return;
    }
    
    // Allocate device memory
    uint8_t *d_headers = NULL;
    uint8_t *d_outputs = NULL;
    
    // Allocate memory for input block headers with error check
    err = hipMalloc((void**)&d_headers, headers_size);
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: Failed to allocate device memory for headers (%zu bytes): %s\n", 
                headers_size, hipGetErrorString(err));
        return;
    }
    
    // Allocate memory for output hashes with error check
    err = hipMalloc((void**)&d_outputs, outputs_size);
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: Failed to allocate device memory for outputs (%zu bytes): %s\n",
                outputs_size, hipGetErrorString(err));
        hipFree(d_headers);
        return;
    }
    
    // Copy block headers from host to device
    err = hipMemcpy(d_headers, block_headers, headers_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: Failed to copy headers to device: %s\n",
                hipGetErrorString(err));
        hipFree(d_headers);
        hipFree(d_outputs);
        return;
    }
    
    // Process one header at a time to isolate any issues
    for (uint32_t i = 0; i < num_blocks; i++) {
        const uint8_t* input = d_headers + i * block_header_len;
        uint8_t* output = d_outputs + i * 32;
        
        // Call rinhash_cuda_kernel with device pointers and proper launch configuration
        rinhash_cuda_kernel<<<1, 32>>>(input, block_header_len, output);
        
        // Check for errors after each processing
        err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "CUDA error in block %u: %s\n", i, hipGetErrorString(err));
            hipFree(d_headers);
            hipFree(d_outputs);
            return;
        }
    }
    
    // Synchronize device to ensure all operations are complete
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error during synchronization: %s\n", hipGetErrorString(err));
        hipFree(d_headers);
        hipFree(d_outputs);
        return;
    }
    
    // Copy results back from device to host
    err = hipMemcpy(outputs, d_outputs, outputs_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: Failed to copy results from device: %s\n",
                hipGetErrorString(err));
    }
    
    // Free device memory
    hipFree(d_headers);
    hipFree(d_outputs);
}

// Main RinHash function that would be called from outside
extern "C" void RinHash(
    const uint32_t* version,
    const uint32_t* prev_block,
    const uint32_t* merkle_root,
    const uint32_t* timestamp,
    const uint32_t* bits,
    const uint32_t* nonce,
    uint8_t* output
) {
    uint8_t block_header[80]; // Standard block header size
    size_t block_header_len;
    
    // Convert block header to bytes
    blockheader_to_bytes(
        version,
        prev_block,
        merkle_root,
        timestamp,
        bits,
        nonce,
        block_header,
        &block_header_len
    );
    
    // Calculate RinHash
    rinhash_cuda(block_header, block_header_len, output);
}

// Mining function that tries different nonces
extern "C" void RinHash_mine(
    const uint32_t* version,
    const uint32_t* prev_block,
    const uint32_t* merkle_root,
    const uint32_t* timestamp,
    const uint32_t* bits,
    uint32_t start_nonce,
    uint32_t num_nonces,
    uint32_t* found_nonce,
    uint8_t* target_hash,
    uint8_t* best_hash
) {
    const size_t block_header_len = 80;
    std::vector<uint8_t> block_headers(block_header_len * num_nonces);
    std::vector<uint8_t> hashes(32 * num_nonces);
    
    // Prepare block headers with different nonces
    for (uint32_t i = 0; i < num_nonces; i++) {
        uint32_t current_nonce = start_nonce + i;
        
        // Fill in the common parts of the header
        uint8_t* header = block_headers.data() + i * block_header_len;
        size_t header_len;
        
        blockheader_to_bytes(
            version,
            prev_block,
            merkle_root,
            timestamp,
            bits,
            &current_nonce,
            header,
            &header_len
        );
    }
    
    // Calculate hashes for all nonces
    rinhash_cuda_batch(block_headers.data(), block_header_len, hashes.data(), num_nonces);
    
    // Find the best hash (lowest value)
    memcpy(best_hash, hashes.data(), 32);
    *found_nonce = start_nonce;
    
    for (uint32_t i = 1; i < num_nonces; i++) {
        uint8_t* current_hash = hashes.data() + i * 32;
        
        // Compare current hash with best hash (byte by byte, from most significant to least)
        bool is_better = false;
        for (int j = 0; j < 32; j++) {
            if (current_hash[j] < best_hash[j]) {
                is_better = true;
                break;
            }
            else if (current_hash[j] > best_hash[j]) {
                break;
            }
        }
        
        if (is_better) {
            memcpy(best_hash, current_hash, 32);
            *found_nonce = start_nonce + i;
        }
    }
}
